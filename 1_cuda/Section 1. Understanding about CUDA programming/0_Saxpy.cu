//Github - https://github.com/arer90/Accelerated_Computing.git

/*
	Section 1. Understanding about CUDA programming
	phase   0. Saxpy (= Saxpy.cu)	

	This is made by 'arer90'
*/
#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>


#include <hipblas.h>

using namespace std;

#define MAX 100
#define RANDOMNUM 100

void CPU_Saxpy(float*x, float*y, float alpha) {
	for (int i = 0; i < MAX; i++) {
		y[i] = alpha * x[i] + y[i];
	}
}

__global__ void GPU_Saxpy(float *x, float *y, float alpha) {
	int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if (idx < MAX) {
		y[idx] = alpha * x[idx] + y[idx];
	}
}

void CUBLAS_saxpy(float *alpha, float *beta, float a, float &time) {
	float *cb_alpha, *cb_beta;
	hipEvent_t init, fin;
	hipEventCreate(&init);
	hipEventCreate(&fin);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipError_t(hipMalloc((void**)&cb_alpha, sizeof(float)*MAX));
	hipError_t(hipMalloc((void**)&cb_beta, sizeof(float)*MAX));
	hipblasSetVector(MAX, sizeof(float), alpha, 1, cb_alpha, 1);
	hipblasSetVector(MAX, sizeof(float), beta, 1, cb_beta, 1);

	hipEventRecord(init, 0);

	hipblasSaxpy(handle, MAX, &a, cb_alpha, 1, cb_beta, 1);

	hipEventRecord(fin, 0);
	hipEventSynchronize(fin);
	hipEventElapsedTime(&time, init, fin);

	hipblasGetVector(MAX, sizeof(float), cb_beta, 1, beta, 1);

	hipblasDestroy(handle);
	hipFree(cb_alpha);
	hipFree(cb_beta);

}

int main() {
	clock_t cpustart, cpuend;
	double cpums;
	float *line1, *line2, *cpuline, *gpuline, *cublasline;
	srand((unsigned)time(NULL));
	float rnum = rand() % RANDOMNUM + 1;

	line1 = new float[MAX];
	line2 = new float[MAX];
	cpuline = new float[MAX];
	gpuline = new float[MAX];
	cublasline = new float[MAX];
	for (int i = 0; i < MAX; i++) {
		line1[i] = rand() % RANDOMNUM + 1;
		line2[i] = rand() % RANDOMNUM + 1;
		cpuline[i] = line2[i];
		gpuline[i] = line2[i];
		cublasline[i] = line2[i];
	}
	//====================================== Checking
	cout << "Linear 1 values." << endl;
	for (int i = 0; i < MAX; i++) {
		cout << line1[i] << " ";
	}
	cout << endl;
	
	cout << "Linear 2 values." << endl;
	for (int i = 0; i < MAX; i++) {
		cout << line2[i] << " ";
	}
	cout << endl;
	//======================================
	cpustart = clock();

	CPU_Saxpy(line1, cpuline, rnum);

	cpuend = clock();
	cpums = (double)((double)cpuend-cpustart/CLOCKS_PER_SEC);
	
	cout << endl;
	cout << "CPU result........" << endl;
	for (int i = 0; i < MAX; i++) {
		cout << cpuline[i] << " ";
	}
	cout << endl;

	//======================================

	hipEvent_t start, stop;
	float gpums, cublasms;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	

	float *dev_line1, *dev_line2;

	hipError_t(hipMalloc((void**)&dev_line1, sizeof(float)*MAX));
	hipError_t(hipMalloc((void**)&dev_line2, sizeof(float)*MAX));
	hipError_t(hipMemcpy(dev_line1, line1, sizeof(float)*MAX, hipMemcpyHostToDevice));
	hipError_t(hipMemcpy(dev_line2, line2, sizeof(float)*MAX, hipMemcpyHostToDevice));

	dim3 thread(32);
	dim3 grids((MAX+32)/MAX);
	
	hipEventRecord(start, 0);
	
	GPU_Saxpy<<<grids, thread>>>(dev_line1, dev_line2, rnum);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpums, start, stop);

	hipError_t(hipMemcpy(gpuline, dev_line2, sizeof(float)*MAX, hipMemcpyDeviceToHost));

	cout << endl;
	cout << "GPU result........" << endl;
	for (int i = 0; i < MAX; i++) {
		cout << gpuline[i] << " ";
	}
	cout << endl;

	//======================================
	
	CUBLAS_saxpy(line1, cublasline, rnum, cublasms);

	cout << endl;
	cout << "cuBLAS result........" << endl;
	for (int i = 0; i < MAX; i++) {
		cout << cublasline[i] << " ";
	}
	cout << endl;
	cout << endl;
	//======================================
	cout << fixed;
	cout.precision(7);
	cout << "CPU time duration for Saxpy function : " << cpums << " (ms)." << endl;
	cout << "GPU time duration for Saxpy function : " << gpums << " (ms)." << endl;
	cout << "cuBLAS time duration for Saxpy function : " << cublasms << " (ms)." << endl;

	hipError_t(hipFree(dev_line1));
	hipError_t(hipFree(dev_line2));

	delete[] line1;
	delete[] line2;
	delete[] cpuline;
	delete[] gpuline;
	delete[] cublasline;

	return 0;
}
